

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <hipblas.h>
#include <vector>
#include <math.h>


// Define some error checking macros.
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
   if (stat != hipSuccess) {
      fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
   }
}

#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
   if (stat != HIPBLAS_STATUS_SUCCESS) {
      fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
   }
}

#define curandErrCheck(stat) { curandErrCheck_((stat), __FILE__, __LINE__); }
void curandErrCheck_(hiprandStatus_t stat, const char *file, int line) {
   if (stat != HIPRAND_STATUS_SUCCESS) {
      fprintf(stderr, "cuRand Error: %d %s %d\n", stat, file, line);
   }
}


__global__ void convertFp32ToFp16 (half *out, float *in, int n) {
   int idx = blockDim.x * blockIdx.x + threadIdx.x;
   if (idx < n) {
      out[idx] = in[idx];
   }
}

__global__ void condition_equal(float *in, float *out)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  out[index] = (((int)ceil(abs(((int)((in[index] + 0.001) * 100.0)/100.0) -1)))&&1) ^ 1;

}

int main(int argc, char* argv[]) {
    int MATRIX_M, MATRIX_N, MATRIX_K;
    int size1= atoi(argv[1]); 
  
    MATRIX_M = size1;
    MATRIX_N = size1;
    MATRIX_K = size1;

   float *c_host_cublas;
   hipblasHandle_t cublasHandle;
   
   hipEvent_t startcublas;
   hipEvent_t stopcublas;

   cudaErrCheck(hipEventCreate(&startcublas));
   cudaErrCheck(hipEventCreate(&stopcublas));
   
   cublasErrCheck(hipblasCreate(&cublasHandle));
   
   // Use tensor cores
   cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH));

   //malloc - void *malloc(size_t size) allocates the requested memory and returns a pointer to it.c function
   c_host_cublas = (float*)malloc(MATRIX_M * MATRIX_N * sizeof(float));

   //create arrays with values
   float *h_a = new float[MATRIX_M*MATRIX_N];
   float *h_b = new float[MATRIX_M*MATRIX_N];
   float *h_c = new float[MATRIX_M*MATRIX_N];

   //TODO - create a array with initial values
    for(int i=0; i<MATRIX_M*MATRIX_N;i++){
        h_a[i] = 0;
        h_b[i] = 0;
        h_c[i] = 0;
    }
    float x =1;
    for(int i=MATRIX_M-1; i<MATRIX_M*MATRIX_N;i+=MATRIX_M){
       h_a[i] = (float)(1/(1/(x++)));
    }
    x=1;
    for(int i=MATRIX_M*MATRIX_N-MATRIX_M ;i<MATRIX_M*MATRIX_N;i++) {
        h_b[i] = (float)(1/(x++));
    }

    //printf("\n");
    //printf("Starting join operation, SELECT * FROM table1 NATURAL JOIN table2;");

	
    // printf("\n");
    // printf("\n------------Printing table A------------\n");
    // for(int i=0; i<MATRIX_M*MATRIX_N;i++){
    //     printf("%f ", h_a[i]);
    //     if((i+1)%MATRIX_M == 0){
    //     printf("\n");
    //     }
    // }

    // printf("\n------------Printing table B------------\n");
    // for(int i=0; i<MATRIX_M*MATRIX_N;i++){
    //     printf("%f ", h_b[i]);
    //     if((i+1)%MATRIX_M == 0){
    //     printf("\n");
    //     }
    // }
	

    //allocate memory and copy the values from CPU to GPU
   float *d_a, *d_b, *d_c;
   half *d_a_half, *d_b_half;
   half *h_a_half = new half[MATRIX_M*MATRIX_N];
   half *h_b_half = new half[MATRIX_M*MATRIX_N];


   cudaErrCheck(hipMalloc((void**)&d_a, MATRIX_M * MATRIX_K * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&d_b, MATRIX_K * MATRIX_N * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&d_c, MATRIX_K * MATRIX_N * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&d_a_half, MATRIX_K * MATRIX_N * sizeof(half)));
   cudaErrCheck(hipMalloc((void**)&d_b_half, MATRIX_K * MATRIX_N * sizeof(half)));

   cudaErrCheck(hipMemcpy(d_a, h_a, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyHostToDevice));
   cudaErrCheck(hipMemcpy(d_b, h_b, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyHostToDevice));


   cudaErrCheck(hipMemcpy(d_a_half, h_a_half, MATRIX_M * MATRIX_N * sizeof(half), hipMemcpyHostToDevice));
   cudaErrCheck(hipMemcpy(d_b_half, h_b_half, MATRIX_M * MATRIX_N * sizeof(half), hipMemcpyHostToDevice));
   cudaErrCheck(hipMemcpy(d_c, h_c, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyHostToDevice));
   
  
   convertFp32ToFp16 <<< (MATRIX_M * MATRIX_K + 255) / 256, 256 >>> (d_a_half, d_a, MATRIX_M*MATRIX_N);
   convertFp32ToFp16 <<< (MATRIX_M * MATRIX_K + 255) / 256, 256  >>> (d_b_half, d_b, MATRIX_M*MATRIX_N);
   
   float alpha = 1.0f;
   float beta = 0.0f;

  // printf("\nM = %d, N = %d, K = %d. alpha = %f, beta = %f\n\n", MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta);

     // Now using cuBLAS
   //printf("------------Running with cuBLAS------------n");

   float cublasTime1;
   float cublasTime2;

   cudaErrCheck(hipEventRecord(startcublas));
   cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                MATRIX_M, MATRIX_N, MATRIX_K, 
                &alpha,
                d_b_half, HIP_R_16F, MATRIX_M,
                d_a_half, HIP_R_16F, MATRIX_K,
                &beta, 
                d_c, HIP_R_32F, MATRIX_M,
                HIP_R_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP));

   	cudaErrCheck(hipEventRecord(stopcublas));
    cudaErrCheck(hipEventSynchronize(stopcublas));
    cudaErrCheck(hipEventElapsedTime(&cublasTime1, startcublas, stopcublas));
    //printf("Matrix multiplication took %fms\n", cublasTime1);
	  
   //Checking Results
   //printf("\n------------Checking results------------\n");
    //   cudaErrCheck(cudaMemcpy(c_host_cublas, d_c, MATRIX_M * MATRIX_N * sizeof(float), cudaMemcpyDeviceToHost));
    
    //    for (int i = 0; i < MATRIX_M*MATRIX_N; i++) {
    //        printf("%f ", c_host_cublas[i]);
    //        if((i+1)%MATRIX_M== 0){
    //          printf("\n");
    //        }
    //    }


	float *d_g;
	float size = MATRIX_M*MATRIX_N * sizeof( float );
	float  *m = (float *)malloc( size );

	cudaErrCheck(hipMalloc( (void **) &d_g, size ));
	cudaErrCheck(hipMemcpy( d_g, m, size, hipMemcpyHostToDevice ));



	/* launch the kernel on the GPU */
	/* insert the launch parameters to launch the kernel properly using blocks and threads */ 

	cudaErrCheck(hipEventRecord(startcublas));
	condition_equal<<< (MATRIX_M * MATRIX_K + 255) / 256, 256>>>(d_c,d_g);
	cudaErrCheck(hipEventRecord(stopcublas));
	
    cudaErrCheck(hipEventSynchronize(stopcublas));
    cudaErrCheck(hipEventElapsedTime(&cublasTime2, startcublas, stopcublas));
    //printf("Bit flipping took %fms\n", cublasTime2);
	printf("%fms", cublasTime1 + cublasTime2);


	cudaErrCheck(hipDeviceSynchronize());

	/* copy result back to host */
	/* fix the parameters needed to copy data back to the host */
	cudaErrCheck(hipMemcpy(m, d_g, size, hipMemcpyDeviceToHost ));
	





    //printf("\n------------Result of JOIN operation -- Printing top 15 results------------\n");
	// for (int i = 0; i < MATRIX_M*MATRIX_N; i++) {
    //    int v2 = m[i];
    //    printf("%d ", v2);
    //    //if(v2==1)
    //      //printf("Merge Row %d of Table 1 and Row %d of Table 2 \n",(int)(i/4), i%4 );
    //     //printf("%f %f %f %f\n", v2*original_table[i] , v2*original_table[i+4], v2*original_table[i+8], v2*original_table[i+12]);
    //    }

	/* clean up */

	free(m);
	hipFree( d_g );

   cudaErrCheck(hipEventDestroy(startcublas));             
   cudaErrCheck(hipEventDestroy(stopcublas));
   
   //free the data
   cudaErrCheck(hipFree(d_a));
   cudaErrCheck(hipFree(d_b));
   cudaErrCheck(hipFree(d_c));
   

   cudaErrCheck(hipDeviceReset());
   return 0;
}