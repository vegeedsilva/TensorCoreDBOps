



#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <hipblas.h>
#include <vector>
#include <math.h>


// Define some error checking macros.
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
   if (stat != hipSuccess) {
      fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
   }
}

#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
   if (stat != HIPBLAS_STATUS_SUCCESS) {
      fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
   }
}

#define curandErrCheck(stat) { curandErrCheck_((stat), __FILE__, __LINE__); }
void curandErrCheck_(hiprandStatus_t stat, const char *file, int line) {
   if (stat != HIPRAND_STATUS_SUCCESS) {
      fprintf(stderr, "cuRand Error: %d %s %d\n", stat, file, line);
   }
}


#define MATRIX_M 512
#define MATRIX_N 512
#define MATRIX_K 512


__global__ void convertFp32ToFp16 (half *out, float *in, int n) {
   int idx = blockDim.x * blockIdx.x + threadIdx.x;
   if (idx < n) {
      out[idx] = in[idx];
   }
}

__global__ void condition_equal(float *in, float *out)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
  out[index] = (((int)ceil(abs(in[index]-1)))&&1) ^ 1;
}

__global__ void condition_greaterthan(float *in, float *out)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
  out[index] = ((int)(ceil(in[index]-1)+ abs(ceil(in[index]-1)))) && 1;
}

__global__ void condition_lessthan(float *in, float *out)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
  out[index] = ((int)(floor(in[index]-1) - abs(floor(in[index]-1)))) && 1;
}

int main(int argc, char* argv[]) {

   float *c_host_cublas;
   int condition;
   hipblasHandle_t cublasHandle;
   
   hipEvent_t startcublas;
   hipEvent_t stopcublas;

   cudaErrCheck(hipEventCreate(&startcublas));
   cudaErrCheck(hipEventCreate(&stopcublas));
   
   cublasErrCheck(hipblasCreate(&cublasHandle));
   
   // Use tensor cores
   //cublasErrCheck(cublasSetMathMode(cublasHandle, CUBLAS_TENSOR_OP_MATH));

   //malloc - void *malloc(size_t size) allocates the requested memory and returns a pointer to it.c function
   c_host_cublas = (float*)malloc(MATRIX_M * MATRIX_N * sizeof(float));

    printf("\n");
    printf("\n------------Original table ------------\n");
    
   float *original_table = new float[MATRIX_M*MATRIX_N];
   
   for(int i=0; i<MATRIX_M*MATRIX_N;i++){
            original_table[i] = i;
            //if(i%MATRIX_M==0)
                //printf("Column %d printing ---->", (i/MATRIX_M)+1);
            //printf(" %d", i);
            //if((i+1)%MATRIX_M == 0){ 
            //printf("\n");
            //}
            
    }

   
   //create arrays with values
   float *h_a = new float[MATRIX_M*MATRIX_N];
   float *h_b = new float[MATRIX_M*MATRIX_N];
   float *h_c = new float[MATRIX_M*MATRIX_N];

   //TODO - create a array with initial values
    for(int i=0; i<MATRIX_M*MATRIX_N;i++){
        h_a[i] = 0;
        h_b[i] = 0;
        h_c[i] = 0;
    }
	    
    for(int i=0;i<MATRIX_M;i++) {
        h_a[i] = original_table[i];
        h_b[i] = 0.5;
    }

    printf("\n");
    printf("Starting select operation, SELECT * FROM table WHERE A ? 2 ");
    printf("\n\n Enter 1 for equality condition\n 2 for less than condition\n 3 for greater than condition\n");
    scanf("%d", &condition);
    printf("\n");
    //printf("\n------------Printing table A------------\n");
    /*for(int i=0; i<MATRIX_M*MATRIX_N;i++){
        //printf("%f ", h_a[i]);
        if((i+1)%MATRIX_M == 0){
        printf("\n");
        }
    }

    printf("\n------------Printing table B------------\n");
    for(int i=0; i<MATRIX_M*MATRIX_N;i++){
        //printf("%f ", h_b[i]);
        if((i+1)%MATRIX_N == 0){
        printf("\n");
        }
    }*/


   //allocate memory and copy the values from CPU to GPU
   float *d_a, *d_b, *d_c;
   half *d_a_half, *d_b_half;
   
   half *h_a_half = new half[MATRIX_M*MATRIX_N];
   half *h_b_half = new half[MATRIX_M*MATRIX_N];


   cudaErrCheck(hipMalloc((void**)&d_a, MATRIX_M * MATRIX_K * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&d_b, MATRIX_K * MATRIX_N * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&d_c, MATRIX_K * MATRIX_N * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&d_a_half, MATRIX_K * MATRIX_N * sizeof(half)));
   cudaErrCheck(hipMalloc((void**)&d_b_half, MATRIX_K * MATRIX_N * sizeof(half)));

   cudaErrCheck(hipMemcpy(d_a, h_a, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyHostToDevice));
   cudaErrCheck(hipMemcpy(d_b, h_b, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyHostToDevice));
   cudaErrCheck(hipMemcpy(d_a_half, h_a_half, MATRIX_M * MATRIX_N * sizeof(half), hipMemcpyHostToDevice));
   cudaErrCheck(hipMemcpy(d_b_half, h_b_half, MATRIX_M * MATRIX_N * sizeof(half), hipMemcpyHostToDevice));
   cudaErrCheck(hipMemcpy(d_c, h_c, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyHostToDevice));
   
  
  
   convertFp32ToFp16 <<< (MATRIX_M * MATRIX_K + 255) / 256, 256 >>> (d_a_half, d_a, MATRIX_M*MATRIX_N);
   convertFp32ToFp16 <<< (MATRIX_M * MATRIX_K + 255) / 256, 256  >>> (d_b_half, d_b, MATRIX_M*MATRIX_N);
   
   float alpha = 1.0f;
   float beta = 1.0f;

   printf("\nM = %d, N = %d, K = %d. alpha = %f, beta = %f\n\n", MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta);

   float cublasTime1;
   float cublasTime2;

     // Now using cuBLAS
   printf("------------Running with cuBLAS------------n");
   cudaErrCheck(hipEventRecord(startcublas));
   cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                MATRIX_M, MATRIX_N, MATRIX_K, 
                &alpha,
                d_a_half, HIP_R_16F, MATRIX_M,
                d_b_half, HIP_R_16F, MATRIX_K,
                &beta, 
                d_c, HIP_R_32F, MATRIX_M,
                HIP_R_32F,  HIPBLAS_GEMM_DEFAULT));
   	cudaErrCheck(hipEventRecord(stopcublas));
    cudaErrCheck(hipEventSynchronize(stopcublas));
    cudaErrCheck(hipEventElapsedTime(&cublasTime1, startcublas, stopcublas));
    printf("Matrix multiplication took %fms\n", cublasTime1);

   // Checking Results
   printf("\n------------Checking results------------\n");
   cudaErrCheck(hipMemcpy(c_host_cublas, d_c, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToHost));
   
  //  for (int i = 0; i < MATRIX_M ; i++) {
  //      printf("%f ", c_host_cublas[i]);
  //      if((i+1)%MATRIX_M== 0){
  //        printf("\n");
  //      }
  //  }

    //----------------------------------------------------

	float *d_g;
	float size = MATRIX_M*MATRIX_N * sizeof( float );

	cudaErrCheck(hipMalloc( (void **) &d_g, size ));
	cudaErrCheck(hipMemcpy( d_g, h_c, size, hipMemcpyHostToDevice ));

    cudaErrCheck(hipEventRecord(startcublas));

	/* launch the kernel on the GPU */
	/* insert the launch parameters to launch the kernel properly using blocks and threads */ 
	  if(condition == 1)
		  condition_equal<<< 1, MATRIX_M >>>(d_c,d_g);
	  else if(condition == 2)
		condition_lessthan<<< 1, MATRIX_M>>>(d_c,d_g);
	  else
		 condition_greaterthan<<< 1, MATRIX_M>>>(d_c,d_g);

    cudaErrCheck(hipEventRecord(stopcublas));
    cudaErrCheck(hipEventSynchronize(stopcublas));
    cudaErrCheck(hipEventElapsedTime(&cublasTime2, startcublas, stopcublas));
    printf("Bit flipping took %fms\n", cublasTime2);
	printf("Total time: %fms\n", cublasTime1 + cublasTime2);
	cudaErrCheck(hipDeviceSynchronize());

	/* copy result back to host */
	/* fix the parameters needed to copy data back to the host */
	cudaErrCheck(hipMemcpy(h_c, d_g, size, hipMemcpyDeviceToHost ));



    printf("\n------------Result of  SELECT operation --Printing top 15 results------------\n");
	for (int i = 0; i < 16; i++) {
       int v2 = h_c[i];
       //printf("%d ",v2);
       //printf("%f %f %f %f\n", v2*original_table[i] , v2*original_table[i+4], v2*original_table[i+8], v2*original_table[i+12]);
      }

	/* clean up */

	hipFree( d_g );

    //------------------------------------
   
      printf("\n\n------------Results verified: cublas------------\n\n");
      float cublasTime;
      //cudaDeviceSynchronize - CPU to wait until the kernel is done before it accesses the results (because CUDA kernel launches don’t block the calling CPU thread
      cudaErrCheck(hipEventSynchronize(stopcublas));
      cudaErrCheck(hipEventElapsedTime(&cublasTime, startcublas, stopcublas));
      printf("cublas took %fms\n", cublasTime);

   cudaErrCheck(hipEventDestroy(startcublas));             
   cudaErrCheck(hipEventDestroy(stopcublas));
   
   //free the data
   cudaErrCheck(hipFree(d_a));
   cudaErrCheck(hipFree(d_b));
   cudaErrCheck(hipFree(d_c));
   
   cudaErrCheck(hipDeviceReset());
   return 0;
}


