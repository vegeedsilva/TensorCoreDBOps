

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <hipblas.h>
#include <vector>
#include <math.h>


// Define some error checking macros.
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
   if (stat != hipSuccess) {
      fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
   }
}

#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
   if (stat != HIPBLAS_STATUS_SUCCESS) {
      fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
   }
}

#define curandErrCheck(stat) { curandErrCheck_((stat), __FILE__, __LINE__); }
void curandErrCheck_(hiprandStatus_t stat, const char *file, int line) {
   if (stat != HIPRAND_STATUS_SUCCESS) {
      fprintf(stderr, "cuRand Error: %d %s %d\n", stat, file, line);
   }
}

__global__ void condition_equal(float *in, float *out, int selectCondition)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(in[index] == selectCondition)  {
    out[index] = 1;
  }
  else {
    out[index] = 0;
  }
}

__global__ void condition_greaterthan(float *in, float *out, int selectCondition)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(in[index] > selectCondition)  {
    out[index] = 1;
  }
  else {
    out[index] = 0;
  }
}

__global__ void condition_lessthan(float *in, float *out, int selectCondition)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(in[index] < selectCondition)  {
    out[index] = 1;
  }
  else {
    out[index] = 0;
  }
}

int main(int argc, char* argv[]) {

   int condition = 1;
   hipblasHandle_t cublasHandle;
   hipEvent_t startcublas;
   hipEvent_t stopcublas;

   int selectCondition = atoi(argv[2]);

   long long MATRIX_M, MATRIX_N, MATRIX_K;
   MATRIX_M = atoi(argv[1]);
//    printf("Enter M size");
//    scanf("%lld", &MATRIX_M);

   MATRIX_N = MATRIX_M;
   MATRIX_K = MATRIX_M;

   cudaErrCheck(hipEventCreate(&startcublas));
   cudaErrCheck(hipEventCreate(&stopcublas));
   cublasErrCheck(hipblasCreate(&cublasHandle));

   float *original_table = new float[MATRIX_M];
   for(int i=0; i<MATRIX_M;i++){
            original_table[i] = i;
            /*
            if(i%MATRIX_M==0)
                printf("Column %d printing ---->", (i/MATRIX_M)+1);
            printf(" %d", i);
            if((i+1)%MATRIX_M == 0){ 
            printf("\n"); 
             }*/
            
    }

   //create arrays with values
   float *h_a = new float[MATRIX_M];
   float *h_c = new float[MATRIX_M];

   //TODO - create a array with initial values
    for(int i=0; i<MATRIX_M;i++){
        h_a[i] = 0;
        h_c[i] = 0;
    }
    
    for(int i=0;i<MATRIX_M;i++) {
        h_a[i] = original_table[i];
    }

    //allocate memory and copy the values from CPU to GPU
   float *d_a, *d_c;
    float  *m;
    float size = MATRIX_M* sizeof( float );
    m = (float *)malloc( size );
   cudaErrCheck(hipMalloc((void**)&d_a, MATRIX_M  * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&d_c, MATRIX_K  * sizeof(float)));


   cudaErrCheck(hipMemcpy(d_a, h_a, MATRIX_M * sizeof(float), hipMemcpyHostToDevice));
   cudaErrCheck(hipMemcpy(d_c, h_c, MATRIX_M * sizeof(float), hipMemcpyHostToDevice));
   
  	cudaErrCheck(hipEventRecord(startcublas));

	  if(condition == 1)
      condition_equal<<< (MATRIX_M + 255) / 256, 256 >>>(d_a, d_c, selectCondition);
	  else if(condition == 2)
		condition_lessthan<<<  (MATRIX_M + 255) / 256, 256 >>>(d_a,d_c, selectCondition);
	  else
		 condition_greaterthan<<<  (MATRIX_M + 255) / 256, 256 >>>(d_a,d_c, selectCondition);

    float cublasTime;
	cudaErrCheck(hipEventRecord(stopcublas));
     //cudaDeviceSynchronize - CPU to wait until the kernel is done before it accesses the results (because CUDA kernel launches don’t block the calling CPU thread
      cudaErrCheck(hipEventSynchronize(stopcublas));
      cudaErrCheck(hipEventElapsedTime(&cublasTime, startcublas, stopcublas));
      printf("%fms", cublasTime);
    cudaErrCheck(hipDeviceSynchronize());

	/* copy result back to host */
	/* fix the parameters needed to copy data back to the host */
	cudaErrCheck(hipMemcpy(m, d_c, size, hipMemcpyDeviceToHost ));

	for (int i = 0; i < 16; i++) {
       int v2 = m[i];
       //printf("%d ",v2);
       //printf("%f %f %f %f\n", v2*original_table[i] , v2*original_table[i+4], v2*original_table[i+8], v2*original_table[i+12]);
    }

	/* clean up */

	free(m);

    //------------------------------------

      //cudaDeviceSynchronize - CPU to wait until the kernel is done before it accesses the results (because CUDA kernel launches don’t block the calling CPU thread
      cudaErrCheck(hipEventSynchronize(stopcublas));
      cudaErrCheck(hipEventElapsedTime(&cublasTime, startcublas, stopcublas));

   cudaErrCheck(hipEventDestroy(startcublas));             
   cudaErrCheck(hipEventDestroy(stopcublas));
   
   //free the data
   cudaErrCheck(hipFree(d_a));
   cudaErrCheck(hipFree(d_c));
   

   cudaErrCheck(hipDeviceReset());
   return 0;
}


