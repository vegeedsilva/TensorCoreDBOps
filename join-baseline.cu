

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <hipblas.h>
#include <vector>
#include <math.h>


// Define some error checking macros.
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
   if (stat != hipSuccess) {
      fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
   }
}

#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
   if (stat != HIPBLAS_STATUS_SUCCESS) {
      fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
   }
}

#define curandErrCheck(stat) { curandErrCheck_((stat), __FILE__, __LINE__); }
void curandErrCheck_(hiprandStatus_t stat, const char *file, int line) {
   if (stat != HIPRAND_STATUS_SUCCESS) {
      fprintf(stderr, "cuRand Error: %d %s %d\n", stat, file, line);
   }
}

__global__ void convertFp32ToFp16 (half *out, float *in, int n) {
   int idx = blockDim.x * blockIdx.x + threadIdx.x;
   if (idx < n) {
      out[idx] = in[idx];
   }
}

__global__ void condition_equal(float *table1, float *table2, float *out, long MATRIX_M, long MATRIX_N)
{
  int index1 = blockIdx.x * blockDim.x + threadIdx.x;
  int index2= blockIdx.y * blockDim.y + threadIdx.y;
  if (index1>=MATRIX_M || index2>=MATRIX_N)
        return;
 
  if(table1[index1] == table2[index2]) {
    out[index1*MATRIX_M+index2] = 1;
  } else {
      out[index1*MATRIX_M+index2] =  0;
  }
}

int main(int argc, char* argv[]) {
   long long MATRIX_M, MATRIX_N, MATRIX_K;
   long long size1= atoi(argv[1]); 
 
   MATRIX_M = size1;
   MATRIX_N = size1;
   MATRIX_K = size1;

   float *c_host_cublas;
   hipblasHandle_t cublasHandle;
   
   hipEvent_t startcublas;
   hipEvent_t stopcublas;

   cudaErrCheck(hipEventCreate(&startcublas));
   cudaErrCheck(hipEventCreate(&stopcublas));
   
   cublasErrCheck(hipblasCreate(&cublasHandle));
   
   // Use tensor cores
   //cublasErrCheck(cublasSetMathMode(cublasHandle, CUBLAS_TENSOR_OP_MATH));

   //malloc - void *malloc(size_t size) allocates the requested memory and returns a pointer to it.c function
   c_host_cublas = (float*)malloc(MATRIX_M * MATRIX_N * sizeof(float));

    //printf("\n");
    //printf("\n------------Table 1------------\n");
    
    float *table1 = new float[MATRIX_M*MATRIX_N];
    float *table2 = new float[MATRIX_M*MATRIX_N];

    for(int i=1; i<=MATRIX_M*MATRIX_N;i++){
            table1[i-1] = i;
             
    }
  
   //printf("\n------------Table 2 ------------\n");
   for(int i=1; i<=MATRIX_M*MATRIX_N;i++){
            table2[i-1] = i;
            
    }

   //create arrays with values
   float *h_a = new float[MATRIX_M*MATRIX_N];
   float *h_b = new float[MATRIX_M*MATRIX_N];
   float *h_c = new float[MATRIX_M*MATRIX_N];

   //TODO - create a array with initial values
    for(int i=0; i<MATRIX_M*MATRIX_N;i++){
        h_a[i] = 0;
        h_b[i] = 0;
        h_c[i] = 0;
    }
    
    for(int i=0;i<MATRIX_M;i++) {
        h_a[i] = table1[i];
        h_b[i] = table2[i];
    }

    //printf("\n");
    //printf("Starting join operation, SELECT * FROM table1 NATURAL JOIN table2;");
    //printf("\nM = %d, N = %d, K = %d.", MATRIX_M, MATRIX_N, MATRIX_K);


    //allocate memory and copy the values from CPU to GPU
   float *d_a, *d_b, *d_c;
   half *d_a_half, *d_b_half;
   half *h_a_half = new half[MATRIX_M*MATRIX_N];
   half *h_b_half = new half[MATRIX_M*MATRIX_N];


   cudaErrCheck(hipMalloc((void**)&d_a, MATRIX_M * MATRIX_K * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&d_b, MATRIX_K * MATRIX_N * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&d_c, MATRIX_K * MATRIX_N * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&d_a_half, MATRIX_K * MATRIX_N * sizeof(half)));
   cudaErrCheck(hipMalloc((void**)&d_b_half, MATRIX_K * MATRIX_N * sizeof(half)));

   cudaErrCheck(hipMemcpy(d_a, h_a, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyHostToDevice));
   cudaErrCheck(hipMemcpy(d_b, h_b, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyHostToDevice));


   cudaErrCheck(hipMemcpy(d_a_half, h_a_half, MATRIX_M * MATRIX_N * sizeof(half), hipMemcpyHostToDevice));
   cudaErrCheck(hipMemcpy(d_b_half, h_b_half, MATRIX_M * MATRIX_N * sizeof(half), hipMemcpyHostToDevice));
   cudaErrCheck(hipMemcpy(d_c, h_c, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyHostToDevice));
   
  
   convertFp32ToFp16 <<< (MATRIX_M * MATRIX_K + 255) / 256, MATRIX_M*MATRIX_N >>> (d_a_half, d_a, MATRIX_M * MATRIX_N);
   convertFp32ToFp16 <<< (MATRIX_M * MATRIX_K + 255) / 256, MATRIX_M*MATRIX_N >>> (d_b_half, d_b, MATRIX_M * MATRIX_N);


    float  *m;
	float *d_g;
	float size = MATRIX_M*MATRIX_N * sizeof( float );
	m = (float *)malloc( size );

	cudaErrCheck(hipMalloc( (void **) &d_g, size ));
    cudaErrCheck(hipMemcpy( d_g, m, size, hipMemcpyHostToDevice ));

	float time;
    cudaErrCheck(hipEventRecord(startcublas));

	/* launch the kernel on the GPU */
	/* insert the launch parameters to launch the kernel properly using blocks and threads */ 
	  dim3 block(MATRIX_M,MATRIX_N);
	  dim3 grid ((MATRIX_M+MATRIX_N-1)/MATRIX_M, (MATRIX_M+MATRIX_N-1)/MATRIX_M  );


	condition_equal<<< grid, block >>>(d_a, d_b, d_g, MATRIX_M, MATRIX_N);

	cudaErrCheck(hipEventRecord(stopcublas));
	cudaErrCheck(hipEventSynchronize(stopcublas));

	cudaErrCheck(hipEventElapsedTime(&time, startcublas, stopcublas));
	cudaErrCheck(hipDeviceSynchronize());

	printf("%fms", time);


	/* copy result back to host */
	/* fix the parameters needed to copy data back to the host */
	cudaErrCheck(hipMemcpy(m, d_g, size, hipMemcpyDeviceToHost ));


     //printf("\n------------Result of JOIN operation ------------\n");
	for (int i = 0; i < MATRIX_M*MATRIX_N; i++) {
       int v2 = m[i];
       //printf("%d ", v2);
       //if(v2==1)
      //printf("Merge Row %d of Table 1 and Row %d of Table 2 \n",(int)(i/4), i%4 );
        //printf("%f %f %f %f\n", v2*original_table[i] , v2*original_table[i+4], v2*original_table[i+8], v2*original_table[i+12]);
       }

	/* clean up */

	free(m);
	hipFree( d_g );

    //------------------------------------
   
      //printf("\n\n------------Results verified: cublas------------\n\n");
      float cublasTime;
      //cudaDeviceSynchronize - CPU to wait until the kernel is done before it accesses the results (because CUDA kernel launches don’t block the calling CPU thread
      cudaErrCheck(hipEventSynchronize(stopcublas));
      cudaErrCheck(hipEventElapsedTime(&cublasTime, startcublas, stopcublas));
      //printf("cublas took %fms\n", cublasTime);

   cudaErrCheck(hipEventDestroy(startcublas));             
   cudaErrCheck(hipEventDestroy(stopcublas));
   
   //free the data
   cudaErrCheck(hipFree(d_a));
   cudaErrCheck(hipFree(d_b));
   cudaErrCheck(hipFree(d_c));
   
   free(c_host_cublas);

   cudaErrCheck(hipDeviceReset());
   return 0;
}